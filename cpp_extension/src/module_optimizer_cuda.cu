#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>

/// @brief GPU配置信息结构体
struct GpuConfig
{
    int max_threads_per_block;    // 每个block最大线程数
    int max_blocks_per_sm;        // 每个SM最大block数
    int multiprocessor_count;     // SM数量
    int max_grid_size;            // 最大grid大小
    size_t global_memory;         // 全局内存大小
    int compute_capability_major; // 计算能力主版本
    int compute_capability_minor; // 计算能力次版本

    // 计算得出的优化参数
    int optimal_block_size;       // 优化的block大小
    int optimal_grid_size;        // 优化的grid大小
    long long optimal_batch_size; // 优化的batch大小
};

/// @brief 属性阈值常量数组
__constant__ int D_ATTR_THRESHOLDS[6] = {1, 4, 8, 12, 16, 20};
/// @brief 基础属性战斗力常量数组
__constant__ int D_BASIC_POWER_VALUES[6] = {7, 14, 29, 44, 167, 254};
/// @brief 特殊属性战斗力常量数组
__constant__ int D_SPECIAL_POWER_VALUES[6] = {14, 29, 59, 89, 298, 448};
/// @brief 特殊属性ID常量数组
__constant__ int D_SPECIAL_ATTRS[8] = {2104, 2105, 2204, 2205, 2404, 2405, 2406, 2304};
/// @brief 总属性战斗力映射表
/// @details 从0到120的属性总值对应的战斗力映射
__constant__ int D_TOTAL_ATTR_POWER_VALUES[121] = {
    0, 5, 11, 17, 23, 29, 34, 40, 46, 52, 58, 64, 69, 75, 81, 87, 93, 99, 104, 110, 116,
    122, 128, 133, 139, 145, 151, 157, 163, 168, 174, 180, 186, 192, 198, 203, 209, 215, 221, 227, 233,
    238, 244, 250, 256, 262, 267, 273, 279, 285, 291, 297, 302, 308, 314, 320, 326, 332, 337, 343, 349,
    355, 361, 366, 372, 378, 384, 390, 396, 401, 407, 413, 419, 425, 431, 436, 442, 448, 454, 460, 466,
    471, 477, 483, 489, 495, 500, 506, 512, 518, 524, 530, 535, 541, 547, 553, 559, 565, 570, 576, 582,
    588, 594, 599, 605, 611, 617, 623, 629, 634, 640, 646, 652, 658, 664, 669, 675, 681, 687, 693, 699};

/// @brief 计算模组组合的战斗力
/// @param combo 模组组合索引数组
/// @param attr_ids 所有模组的属性ID数组
/// @param attr_values 所有模组的属性值数组
/// @param attr_counts 每个模组的属性数量数组
/// @param offsets 每个模组在属性数组中的偏移量
/// @param target_attrs 目标属性ID数组
/// @param target_count 目标属性数量
/// @param exclude_attrs 排除属性ID数组
/// @param exclude_count 排除属性数量
/// @return 战力
__device__ int CalculatePowerGpu(
    const int *combo,
    const int *attr_ids,
    const int *attr_values,
    const int *attr_counts,
    const int *offsets,
    const int *target_attrs,
    int target_count,
    const int *exclude_attrs,
    int exclude_count)
{
    int aggregated_ids[20];
    int aggregated_values[20];
    int agg_count = 0;
    int total_attr_value = 0;

    for (int m = 0; m < 4; ++m)
    {
        int module_idx = combo[m];
        int start_offset = offsets[module_idx];
        int attr_cnt = attr_counts[module_idx];

        for (int i = 0; i < attr_cnt; ++i)
        {
            int attr_id = attr_ids[start_offset + i];
            int attr_value = attr_values[start_offset + i];

            total_attr_value += attr_value;

            int found_idx = -1;
            for (int j = 0; j < agg_count; ++j)
            {
                if (aggregated_ids[j] == attr_id)
                {
                    found_idx = j;
                    break;
                }
            }

            if (found_idx >= 0)
            {
                aggregated_values[found_idx] += attr_value;
            }
            else
            {
                aggregated_ids[agg_count] = attr_id;
                aggregated_values[agg_count] = attr_value;
                agg_count++;
            }
        }
    }

    int threshold_power = 0;

    for (int i = 0; i < agg_count; ++i)
    {
        int attr_id = aggregated_ids[i];
        int attr_value = aggregated_values[i];

        int max_level = 0;
        for (int j = 0; j < 6; ++j)
        {
            if (attr_value >= D_ATTR_THRESHOLDS[j])
            {
                max_level = j + 1;
            }
        }

        if (max_level > 0)
        {
            bool is_special = false;
            for (int j = 0; j < 8; ++j)
            {
                if (attr_id == D_SPECIAL_ATTRS[j])
                {
                    is_special = true;
                    break;
                }
            }

            int base_power = is_special ? D_SPECIAL_POWER_VALUES[max_level - 1] : D_BASIC_POWER_VALUES[max_level - 1];
            int power_multiplier = 1;

            for (int j = 0; j < target_count; ++j)
            {
                if (attr_id == target_attrs[j])
                {
                    power_multiplier = 2;
                    break;
                }
            }

            if (power_multiplier != 2)
            {
                for (int j = 0; j < exclude_count; ++j)
                {
                    if (attr_id == exclude_attrs[j])
                    {
                        power_multiplier = 0;
                        break;
                    }
                }
            }

            threshold_power += base_power * power_multiplier;
        }
    }

    int total_attr_power = D_TOTAL_ATTR_POWER_VALUES[total_attr_value];

    return threshold_power + total_attr_power;
}

/// @brief 用于判断是否支持CUDA加速
/// @param data 数据数组指针
/// @param size 数据数组大小
__global__ void TestKernel(int *data, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        data[idx] = idx * 2;
    }
}

/// @brief 计算组合数
/// @param n 总元素数量
/// @param r 选择元素数量
/// @return 组合数
__device__ long long GpuCombinationCount(int n, int r)
{
    if (r > n || r < 0)
        return 0;
    if (r == 0 || r == n)
        return 1;
    if (r > n - r)
        r = n - r;

    long long result = 1;
    for (int i = 0; i < r; ++i)
    {
        result = result * (n - i) / (i + 1);
    }
    return result;
}

/// @brief 根据索引生成第 k 个组合
/// @param n 总元素数量
/// @param r 选择元素数量
/// @param index 组合索引
/// @param combination 组合结果
__device__ void GpuGetCombinationByIndex(int n, int r, long long index, int *combination)
{
    long long remaining = index;

    for (int i = 0; i < r; ++i)
    {
        int start = (i == 0) ? 0 : combination[i - 1] + 1;
        for (int j = start; j < n; ++j)
        {
            long long combinations_after = GpuCombinationCount(n - j - 1, r - i - 1);
            if (remaining < combinations_after)
            {
                combination[i] = j;
                break;
            }
            remaining -= combinations_after;
        }
    }
}

__device__ inline bool GpuNextCombination(int n, int r, int *comb)
{
    for (int pos = r - 1; pos >= 0; --pos)
    {
        int limit = n - r + pos;
        if (comb[pos] < limit)
        {
            ++comb[pos];
            for (int k = pos + 1; k < r; ++k)
            {
                comb[k] = comb[k - 1] + 1;
            }
            return true;
        }
    }
    return false;
}

/// @brief CUDA枚举算子
/// @param attr_ids 所有模组的属性ID数组
/// @param attr_values 所有模组的属性值数组
/// @param attr_counts 每个模组的属性数量数组
/// @param offsets 每个模组在属性数组中的偏移量
/// @param module_count 模组总数
/// @param start_combination 起始组合索引
/// @param end_combination 结束组合索引
/// @param target_attrs 目标属性ID数组
/// @param target_count 目标属性数量
/// @param exclude_attrs 排除属性ID数组
/// @param exclude_count 排除属性数量
/// @param min_attr_ids 最小属性需求ID数组
/// @param min_attr_values 最小属性需求值数组
/// @param min_attr_count 最小属性需求数量
/// @param scores 输出参数: 计算得到的战斗力数组
/// @param indices 输出参数: 打包的模组索引数组
__global__ void GpuEnumerationKernel(
    const int *attr_ids,
    const int *attr_values,
    const int *attr_counts,
    const int *offsets,
    int module_count,
    long long start_combination,
    long long end_combination,
    const int *target_attrs,
    int target_count,
    const int *exclude_attrs,
    int exclude_count,
    const int *min_attr_ids,
    const int *min_attr_values,
    int min_attr_count,
    int *scores,
    long long *indices)
{
    long long tid = blockIdx.x * blockDim.x + threadIdx.x;
    long long total_threads = gridDim.x * blockDim.x;

    long long S = start_combination;
    long long E = end_combination;
    long long R = E - S;
    if (R <= 0)
        return;

    long long L = (R + total_threads - 1) / total_threads;
    long long seg_start = S + tid * L;
    if (seg_start >= E)
        return;
    long long seg_end = min(seg_start + L, E);

    int combo[4];
    GpuGetCombinationByIndex(module_count, 4, seg_start, combo);

    long long local_offset = 0;
    for (long long combo_idx = seg_start; combo_idx < seg_end; ++combo_idx, ++local_offset)
    {
        if (min_attr_count > 0)
        {
            bool valid = true;
            for (int req_idx = 0; req_idx < min_attr_count; ++req_idx)
            {
                int required_attr_id = min_attr_ids[req_idx];
                int required_min_value = min_attr_values[req_idx];
                int actual_sum = 0;

                for (int m = 0; m < 4; ++m)
                {
                    int module_idx = combo[m];
                    int start_offset = offsets[module_idx];
                    int attr_cnt = attr_counts[module_idx];

                    for (int i = 0; i < attr_cnt; ++i)
                    {
                        int attr_id = attr_ids[start_offset + i];
                        if (attr_id == required_attr_id)
                        {
                            actual_sum += attr_values[start_offset + i];
                        }
                    }
                }

                if (actual_sum < required_min_value)
                {
                    valid = false;
                    break;
                }
            }
            if (!valid)
            {
                if (!GpuNextCombination(module_count, 4, combo))
                    break;
                continue;
            }
        }

        int combat_power = CalculatePowerGpu(
            combo, attr_ids, attr_values, attr_counts, offsets,
            target_attrs, target_count, exclude_attrs, exclude_count);

        long long packed = 0;
        for (int i = 0; i < 4; ++i)
        {
            packed |= ((long long)combo[i] << (i * 16));
        }

        long long output_idx = (seg_start - S) + local_offset;
        scores[output_idx] = combat_power;
        indices[output_idx] = packed;

        if (!GpuNextCombination(module_count, 4, combo))
            break;
    }
}

/// @brief 获取前TOP解
/// @param d_scores 分数数组
/// @param d_indices 索引数组
/// @param total_count 总结果数量
/// @param top_count 需要的TOP结果
void GpuSortTopSolutions(int *d_scores, long long *d_indices, int total_count, int top_count)
{
    thrust::device_ptr<int> scores_ptr(d_scores);
    thrust::device_ptr<long long> indices_ptr(d_indices);
    thrust::sort_by_key(scores_ptr, scores_ptr + total_count, indices_ptr, thrust::greater<int>());
}

/// @brief 获取GPU配置信息
/// @param config 输出的GPU配置信息
/// @return 1表示成功，0表示失败
int GetGpuConfig(GpuConfig *config)
{
    hipError_t err;
    hipDeviceProp_t prop;

    err = hipGetDeviceProperties(&prop, 0);
    if (err != hipSuccess)
    {
        return 0;
    }

    config->max_threads_per_block = prop.maxThreadsPerBlock;
    config->max_blocks_per_sm = prop.maxBlocksPerMultiProcessor;
    config->multiprocessor_count = prop.multiProcessorCount;
    config->max_grid_size = prop.maxGridSize[0];
    config->global_memory = prop.totalGlobalMem;
    config->compute_capability_major = prop.major;
    config->compute_capability_minor = prop.minor;

    return 1;
}

/// @brief 计算优化的GPU执行参数
/// @param config GPU配置信息
/// @param total_combinations 总组合数
void CalculateOptimalParams(GpuConfig *config, long long total_combinations)
{
    // 计算优化的block大小
    if (config->compute_capability_major >= 7)
    {
        // 较新的GPU
        config->optimal_block_size = 512;
    }
    else if (config->compute_capability_major >= 6)
    {
        // Pascal架构
        config->optimal_block_size = 256;
    }
    else
    {
        // 较老的GPU
        config->optimal_block_size = 192;
    }

    // 确保不超过硬件限制
    config->optimal_block_size = min(config->optimal_block_size, config->max_threads_per_block);

    // 计算优化的grid大小
    int total_cores = config->multiprocessor_count * config->max_blocks_per_sm;
    config->optimal_grid_size = min(total_cores * 2, config->max_grid_size);

    // 基于实际工作负载调整
    long long max_concurrent_threads = (long long)config->optimal_grid_size * config->optimal_block_size;
    if (total_combinations < max_concurrent_threads)
    {
        config->optimal_grid_size = (int)((total_combinations + config->optimal_block_size - 1) / config->optimal_block_size);
    }

    // 计算优化的batch大小
    size_t available_memory = config->global_memory * 0.5;
    long long memory_limited_batch = available_memory / (sizeof(int) + sizeof(long long));

    // 基于计算能力的batch大小
    long long compute_limited_batch = max_concurrent_threads * 1000;

    // 取较小值, 但至少10万, 最大500万
    config->optimal_batch_size = max(100000LL, min(memory_limited_batch, compute_limited_batch));
    config->optimal_batch_size = min(config->optimal_batch_size, 5000000LL);
}

/// @brief 用于判断是否支持CUDA加速
/// @return 1表示CUDA可用，0表示CUDA不可用
extern "C" int TestCuda()
{
    int device_count = 0;
    hipError_t err = hipGetDeviceCount(&device_count);

    if (err != hipSuccess || device_count == 0)
    {
        return 0;
    }

    int *d_data;
    const int size = 1024;
    err = hipMalloc(&d_data, size * sizeof(int));
    if (err != hipSuccess)
    {
        return 0;
    }

    dim3 block(256);
    dim3 grid((size + block.x - 1) / block.x);
    TestKernel<<<grid, block>>>(d_data, size);

    err = hipDeviceSynchronize();
    hipFree(d_data);

    return (err == hipSuccess) ? 1 : 0;
}

/// @brief 计算组合数
/// @param n 总元素数量
/// @param r 选择元素数量
/// @return 组合数
long long CpuCombinationCount(int n, int r)
{
    if (r > n || r < 0)
        return 0;
    if (r == 0 || r == n)
        return 1;
    if (r > n - r)
        r = n - r;

    long long result = 1;
    for (int i = 0; i < r; ++i)
    {
        result = result * (n - i) / (i + 1);
    }
    return result;
}

/// @brief 完整的CUDA策略枚举函数
/// @param module_attr_ids 所有模组的属性ID数组
/// @param module_attr_values 所有模组的属性值数组
/// @param module_attr_counts 每个模组的属性数量数组
/// @param module_offsets 每个模组在属性数组中的偏移量
/// @param module_count 模组总数
/// @param total_attrs 总属性数量
/// @param target_attrs 目标属性ID数组
/// @param target_count 目标属性数量
/// @param exclude_attrs 排除属性ID数组
/// @param exclude_count 排除属性数量
/// @param min_attr_ids 最小属性需求ID数组
/// @param min_attr_values 最小属性需求值数组
/// @param min_attr_count 最小属性需求数量
/// @param max_solutions 最大解决方案数量
/// @param result_scores 输出参数：结果分数数组
/// @param result_indices 输出参数：结果模组索引数组
/// @return 成功处理的解决方案数量，0表示失败
extern "C" int GpuStrategyEnumeration(
    const int *module_attr_ids,
    const int *module_attr_values,
    const int *module_attr_counts,
    const int *module_offsets,
    int module_count,
    int total_attrs,
    const int *target_attrs,
    int target_count,
    const int *exclude_attrs,
    int exclude_count,
    const int *min_attr_ids,
    const int *min_attr_values,
    int min_attr_count,
    int max_solutions,
    int *result_scores,
    long long *result_indices)
{
    // 计算要处理的组合数
    long long total_combinations = CpuCombinationCount(module_count, 4);

    // 获取GPU配置并计算优化参数
    GpuConfig gpu_config;
    if (!GetGpuConfig(&gpu_config))
    {
        printf("Failed to get GPU configuration\n");
        return 0;
    }

    CalculateOptimalParams(&gpu_config, total_combinations);

    printf("GPU Configuration:\n");
    printf("  Compute Capability: %d.%d\n", gpu_config.compute_capability_major, gpu_config.compute_capability_minor);
    printf("  Multiprocessors: %d\n", gpu_config.multiprocessor_count);
    printf("  Global Memory: %.1f MB\n", (double)gpu_config.global_memory / (1024 * 1024));
    printf("Optimal Parameters:\n");
    printf("  Block Size: %d\n", gpu_config.optimal_block_size);
    printf("  Grid Size: %d\n", gpu_config.optimal_grid_size);
    printf("  Batch Size: %lld\n", gpu_config.optimal_batch_size);

    // 使用优化的批处理大小
    long long batch_size = gpu_config.optimal_batch_size;

    std::vector<int> global_best_scores(max_solutions, 0);
    std::vector<long long> global_best_indices(max_solutions, 0);
    long long processed = 0;

    // 分配GPU内存
    int *d_attr_ids = nullptr;
    int *d_attr_values = nullptr;
    int *d_attr_counts = nullptr;
    int *d_offsets = nullptr;
    int *d_target_attrs = nullptr;
    int *d_exclude_attrs = nullptr;
    int *d_min_attr_ids = nullptr;
    int *d_min_attr_values = nullptr;
    int *d_scores = nullptr;
    long long *d_indices = nullptr;

    hipError_t err;

    err = hipMalloc(&d_attr_ids, total_attrs * sizeof(int));
    if (err != hipSuccess)
    {
        printf("ERROR: CUDA malloc failed (attr_ids): %s\n", hipGetErrorString(err));
        return 0;
    }

    err = hipMalloc(&d_attr_values, total_attrs * sizeof(int));
    if (err != hipSuccess)
    {
        printf("ERROR: CUDA malloc failed (attr_values): %s\n", hipGetErrorString(err));
        hipFree(d_attr_ids);
        return 0;
    }

    err = hipMalloc(&d_attr_counts, module_count * sizeof(int));
    if (err != hipSuccess)
    {
        printf("ERROR: CUDA malloc failed(attr_counts): %s\n", hipGetErrorString(err));
        hipFree(d_attr_ids);
        hipFree(d_attr_values);
        return 0;
    }

    err = hipMalloc(&d_offsets, module_count * sizeof(int));
    if (err != hipSuccess)
    {
        printf("ERROR: CUDA malloc failed(offsets): %s\n", hipGetErrorString(err));
        hipFree(d_attr_ids);
        hipFree(d_attr_values);
        hipFree(d_attr_counts);
        return 0;
    }

    if (target_count > 0)
    {
        err = hipMalloc(&d_target_attrs, target_count * sizeof(int));
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA malloc failed(target_attrs): %s\n", hipGetErrorString(err));
            goto cleanup;
        }
        err = hipMemcpy(d_target_attrs, target_attrs, target_count * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA memcpy failed(target_attrs): %s\n", hipGetErrorString(err));
            goto cleanup;
        }
    }

    if (exclude_count > 0)
    {
        err = hipMalloc(&d_exclude_attrs, exclude_count * sizeof(int));
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA malloc failed(exclude_attrs): %s\n", hipGetErrorString(err));
            goto cleanup;
        }
        err = hipMemcpy(d_exclude_attrs, exclude_attrs, exclude_count * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA memcpy failed(exclude_attrs): %s\n", hipGetErrorString(err));
            goto cleanup;
        }
    }

    if (min_attr_count > 0)
    {
        err = hipMalloc(&d_min_attr_ids, min_attr_count * sizeof(int));
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA malloc failed(min_attr_ids): %s\n", hipGetErrorString(err));
            goto cleanup;
        }
        err = hipMemcpy(d_min_attr_ids, min_attr_ids, min_attr_count * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA memcpy failed(min_attr_ids): %s\n", hipGetErrorString(err));
            goto cleanup;
        }

        err = hipMalloc(&d_min_attr_values, min_attr_count * sizeof(int));
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA malloc failed(min_attr_values): %s\n", hipGetErrorString(err));
            goto cleanup;
        }
        err = hipMemcpy(d_min_attr_values, min_attr_values, min_attr_count * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA memcpy failed(min_attr_values): %s\n", hipGetErrorString(err));
            goto cleanup;
        }
    }

    err = hipMalloc(&d_scores, batch_size * sizeof(int));
    if (err != hipSuccess)
    {
        printf("ERROR: CUDA malloc failed(scores): %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    err = hipMalloc(&d_indices, batch_size * sizeof(long long));
    if (err != hipSuccess)
    {
        printf("ERROR: CUDA malloc failed(indices): %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    err = hipMemcpy(d_attr_ids, module_attr_ids, total_attrs * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("ERROR: CUDA memcpy failed(attr_ids): %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    err = hipMemcpy(d_attr_values, module_attr_values, total_attrs * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("ERROR: CUDA memcpy failed(attr_values): %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    err = hipMemcpy(d_attr_counts, module_attr_counts, module_count * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("ERROR: CUDA memcpy failed(attr_counts): %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    err = hipMemcpy(d_offsets, module_offsets, module_count * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("ERROR: CUDA memcpy failed(offsets): %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    // 开始处理所有组合
    for (long long batch_start = 0; batch_start < total_combinations; batch_start += batch_size)
    {
        long long current_batch_size = min(batch_size, total_combinations - batch_start);

        // 创建当前结果内存
        err = hipMemset(d_scores, 0, current_batch_size * sizeof(int));
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA memset failed(scores): %s\n", hipGetErrorString(err));
            goto cleanup;
        }

        err = hipMemset(d_indices, 0, current_batch_size * sizeof(long long));
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA memset failed(indices): %s\n", hipGetErrorString(err));
            goto cleanup;
        }

        // 执行kernel
        {
            dim3 block(gpu_config.optimal_block_size);
            int grid_size = min(gpu_config.optimal_grid_size, (int)((current_batch_size + block.x - 1) / block.x));
            dim3 grid(grid_size);

            GpuEnumerationKernel<<<grid, block>>>(
                d_attr_ids, d_attr_values, d_attr_counts, d_offsets,
                module_count, batch_start, batch_start + current_batch_size,
                d_target_attrs, target_count,
                d_exclude_attrs, exclude_count,
                d_min_attr_ids, d_min_attr_values, min_attr_count,
                d_scores, d_indices);

            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("ERROR: CUDA kernel launch failed: %s\n", hipGetErrorString(err));
                goto cleanup;
            }

            err = hipDeviceSynchronize();
            if (err != hipSuccess)
            {
                printf("ERROR: CUDA kernel execution failed: %s\n", hipGetErrorString(err));
                goto cleanup;
            }

            // 排序
            if (current_batch_size >= max_solutions)
            {
                GpuSortTopSolutions(d_scores, d_indices, (int)current_batch_size, max_solutions);
            }
        }

        // 获取当前批次Top解
        int results_to_transfer = min((long long)max_solutions, current_batch_size);

        std::vector<int> batch_scores(results_to_transfer);
        std::vector<long long> batch_indices(results_to_transfer);

        err = hipMemcpy(batch_scores.data(), d_scores, results_to_transfer * sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA result transfer failed(batch_scores): %s\n", hipGetErrorString(err));
            printf("Debug: d_scores=%p, batch_scores.data()=%p, size=%d bytes\n",
                   d_scores, batch_scores.data(), results_to_transfer * (int)sizeof(int));
            goto cleanup;
        }

        err = hipMemcpy(batch_indices.data(), d_indices, results_to_transfer * sizeof(long long), hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            printf("ERROR: CUDA result transfer failed(batch_indices): %s\n", hipGetErrorString(err));
            goto cleanup;
        }

        // 合并当前批次结果到全局TOP
        for (int i = 0; i < results_to_transfer; ++i)
        {
            bool should_insert = false;
            int insert_pos = max_solutions;

            // 查找插入位置
            for (int j = 0; j < max_solutions; ++j)
            {
                if (global_best_scores[j] == 0 || batch_scores[i] > global_best_scores[j])
                {
                    insert_pos = j;
                    should_insert = true;
                    break;
                }
            }

            if (should_insert && insert_pos < max_solutions)
            {
                for (int j = max_solutions - 1; j > insert_pos; --j)
                {
                    global_best_scores[j] = global_best_scores[j - 1];
                    global_best_indices[j] = global_best_indices[j - 1];
                }

                global_best_scores[insert_pos] = batch_scores[i];
                global_best_indices[insert_pos] = batch_indices[i];
            }
        }

        processed += current_batch_size;
    }

    for (int i = 0; i < max_solutions; ++i)
    {
        result_scores[i] = global_best_scores[i];
        result_indices[i] = global_best_indices[i];
    }

    // 清理GPU内存
cleanup:
    if (d_attr_ids)
        hipFree(d_attr_ids);
    if (d_attr_values)
        hipFree(d_attr_values);
    if (d_attr_counts)
        hipFree(d_attr_counts);
    if (d_offsets)
        hipFree(d_offsets);
    if (d_target_attrs)
        hipFree(d_target_attrs);
    if (d_exclude_attrs)
        hipFree(d_exclude_attrs);
    if (d_min_attr_ids)
        hipFree(d_min_attr_ids);
    if (d_min_attr_values)
        hipFree(d_min_attr_values);
    if (d_scores)
        hipFree(d_scores);
    if (d_indices)
        hipFree(d_indices);

    return (err == hipSuccess) ? max_solutions : 0;
}
